#include "hip/hip_runtime.h"
/**
* This file is part of Jetson-SLAM.
*
* Written by Ashish Kumar Indian Institute of Tehcnology, Kanpur, India
* For more information see <https://github.com/ashishkumar822/Jetson-SLAM>
*
* Jetson-SLAM is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* Jetson-SLAM is distributed WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
*/

//// Part of the Middle-end

#include "cuda/orb_matcher.hpp"

#include<hip/hip_runtime.h>
#include<cuda_device_runtime_api.h>
#include<hip/hip_runtime_api.h>

#include<stdio.h>

#include<iostream>

#include<chrono>

namespace orb_cuda{

#define CUDA_NUM_THREADS_PER_BLOCK 512

__global__ void ORB_Search_by_projection_project_on_GPU(int n_threads,
                                                        float* Px, float* Py, float* Pz,
                                                        float* Rcw, float* tcw,
                                                        float fx, float fy, float cx, float cy,
                                                        float minX, float maxX, float minY, float maxY,
                                                        float* u, float* v, float* invz,
                                                        unsigned char* is_valid)
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index <  n_threads)
    {
        unsigned char isvalid = 0;

        const float Pwx = Px[index];
        const float Pwy = Py[index];
        const float Pwz = Pz[index];

        const float Pcx = Pwx * Rcw[0] + Pwy * Rcw[1] + Pwz * Rcw[2] + tcw[0];
        const float Pcy = Pwx * Rcw[3] + Pwy * Rcw[4] + Pwz * Rcw[5] + tcw[1];
        const float Pcz = Pwx * Rcw[6] + Pwy * Rcw[7] + Pwz * Rcw[8] + tcw[2];

        float im_invz = -1;
        float im_u = -1;
        float im_v = -1;

        // Check positive depth
        if(Pcz > 0.0f)
        {
            // Project in image and check it is not outside
            im_invz = 1.0f / Pcz;
            im_u = fx * Pcx * im_invz + cx;
            im_v = fy * Pcy * im_invz + cy;

            if(!(im_u < minX || im_u > maxX || im_v < minY || im_v > maxY))
                isvalid = 1;

        }

        u[index] = im_u;
        v[index] = im_v;
        invz[index] = im_invz;
        is_valid[index] = isvalid;
    }
}


void ORB_Search_by_projection_project_on_frame(int n_points,
                                               float* Px_gpu, float* Py_gpu, float* Pz_gpu,
                                               float* Rcw_gpu, float* tcw_gpu,
                                               float& fx, float& fy, float& cx, float& cy,
                                               float& minX, float& maxX, float& minY, float& maxY,
                                               float* u_gpu, float* v_gpu, float* invz_gpu,
                                               unsigned char* is_valid_gpu)
{

    int n_threads = n_points;

    int CUDA_NUM_BLOCKS = (n_threads + CUDA_NUM_THREADS_PER_BLOCK) / CUDA_NUM_THREADS_PER_BLOCK;

    ORB_Search_by_projection_project_on_GPU<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS_PER_BLOCK>>>(
                                                                                               n_threads,
                                                                                               Px_gpu, Py_gpu, Pz_gpu,
                                                                                               Rcw_gpu, tcw_gpu,
                                                                                               fx, fy, cx, cy,
                                                                                               minX, maxX, minY, maxY,
                                                                                               u_gpu,  v_gpu, invz_gpu,
                                                                                               is_valid_gpu
                                                                                               );

    hipStreamSynchronize(0);
}





__global__ void ORB_compute_descriptor_Distance_GPU(int n_threads,
                                                    int* idx_left, int* idx_right,
                                                    unsigned char*  descriptor_left,
                                                    unsigned char*  descriptor_right,
                                                    int* distance)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index <  n_threads)
    {
        int* d_left  = (int*)(descriptor_left  + idx_left[index] * 32);
        int* d_right = (int*)(descriptor_right + idx_right[index] * 32);

        int dist=0;

        for(int i=0; i<8; i++)
        {
            unsigned  int v = d_left[i] ^ d_right[i];
            v = v - ((v >> 1) & 0x55555555);
            v = (v & 0x33333333) + ((v >> 2) & 0x33333333);
            dist += (((v + (v >> 4)) & 0xF0F0F0F) * 0x1010101) >> 24;
        }

        distance[index] = dist;
    }
}


void ORB_compute_distances(int n_points,
                           int* idx_left, int* idx_right,
                           unsigned char*  descriptor_left,
                           unsigned char*  descriptor_right,
                           int* distance)
{
    int n_threads =  n_points;

    int CUDA_NUM_BLOCKS = (n_threads + CUDA_NUM_THREADS_PER_BLOCK) / CUDA_NUM_THREADS_PER_BLOCK;

    ORB_compute_descriptor_Distance_GPU<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS_PER_BLOCK>>>(
                                                                                           n_threads,
                                                                                           idx_left, idx_right,
                                                                                           descriptor_left,
                                                                                           descriptor_right,
                                                                                           distance
                                                                                           );


    hipStreamSynchronize(0);

//    std::cout << "line " << __LINE__ << " curror =  " << hipGetLastError() << "\n";

}

}

