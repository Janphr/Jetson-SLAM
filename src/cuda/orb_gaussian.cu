#include "hip/hip_runtime.h"
#include "cuda/orb_gpu.hpp"

#include<vector>
#include<opencv2/opencv.hpp>

#include<hip/hip_runtime.h>
#include<cuda_device_runtime_api.h>
#include<hip/hip_runtime_api.h>

#include<chrono>


namespace orb_cuda {


#define CUDA_NUM_THREADS_PER_BLOCK_V2 32

#define CUDA_NUM_THREADS_PER_BLOCK 512


__global__ void imgaussian_GPU(int n_threads,
                               int height, int width,
                               int roiheight, int roiwidth,
                               unsigned char* image_data,
                               int image_pitch,
                               unsigned char* image_data_gaussian,
                               int image_gaussian_pitch,
                               float* gaussian_weights)
{
    int index = blockDim.x * blockIdx.x +  threadIdx.x;

    if(index < n_threads)
    {
        int h = index / roiwidth;
        int w = index % roiwidth;

        int h_im = h + BORDER_SKIP;
        int w_im = w + BORDER_SKIP;

        int count = 0;
        float val = 0;

//#define GAUSSIAN_UNROLL

#ifndef GAUSSIAN_UNROLL

#pragma unroll
        for(int i=-3;i<=3;i++)
        {
            int row_offset = (h_im + i) * image_pitch;
#pragma unroll
            for(int j=-3;j<=3;j++)
            {
                int col_offset = w_im + j;
                val += gaussian_weights[count++] * image_data[row_offset + col_offset];
            }
        }

#else

        int offset = 0;

        offset = (h_im - 3) * image_pitch + w_im;

        val += gaussian_weights[0] * image_data[offset-3];
        val += gaussian_weights[1] * image_data[offset-2];
        val += gaussian_weights[2] * image_data[offset-1];
        val += gaussian_weights[3] * image_data[offset];
        val += gaussian_weights[4] * image_data[offset+1];
        val += gaussian_weights[5] * image_data[offset+2];
        val += gaussian_weights[6] * image_data[offset+3];

        offset = (h_im - 2) * image_pitch + w_im;

        val += gaussian_weights[7] * image_data[offset-3];
        val += gaussian_weights[8] * image_data[offset-2];
        val += gaussian_weights[9] * image_data[offset-1];
        val += gaussian_weights[10] * image_data[offset];
        val += gaussian_weights[11] * image_data[offset+1];
        val += gaussian_weights[12] * image_data[offset+2];
        val += gaussian_weights[13] * image_data[offset+3];


        offset = (h_im - 1) * image_pitch + w_im;

        val += gaussian_weights[14] * image_data[offset-3];
        val += gaussian_weights[15] * image_data[offset-2];
        val += gaussian_weights[16] * image_data[offset-1];
        val += gaussian_weights[17] * image_data[offset];
        val += gaussian_weights[18] * image_data[offset+1];
        val += gaussian_weights[19] * image_data[offset+2];
        val += gaussian_weights[20] * image_data[offset+3];

        offset = (h_im - 0) * image_pitch + w_im;

        val += gaussian_weights[21] * image_data[offset-3];
        val += gaussian_weights[22] * image_data[offset-2];
        val += gaussian_weights[23] * image_data[offset-1];
        val += gaussian_weights[24] * image_data[offset];
        val += gaussian_weights[25] * image_data[offset+1];
        val += gaussian_weights[26] * image_data[offset+2];
        val += gaussian_weights[27] * image_data[offset+3];

        offset = (h_im + 1) * image_pitch + w_im;

        val += gaussian_weights[28] * image_data[offset-3];
        val += gaussian_weights[29] * image_data[offset-2];
        val += gaussian_weights[30] * image_data[offset-1];
        val += gaussian_weights[31] * image_data[offset];
        val += gaussian_weights[32] * image_data[offset+1];
        val += gaussian_weights[33] * image_data[offset+2];
        val += gaussian_weights[34] * image_data[offset+3];

        offset = (h_im + 2) * image_pitch + w_im;

        val += gaussian_weights[35] * image_data[offset-3];
        val += gaussian_weights[36] * image_data[offset-2];
        val += gaussian_weights[37] * image_data[offset-1];
        val += gaussian_weights[38] * image_data[offset];
        val += gaussian_weights[39] * image_data[offset+1];
        val += gaussian_weights[40] * image_data[offset+2];
        val += gaussian_weights[41] * image_data[offset+3];

        offset = (h_im + 3) * image_pitch + w_im;

        val += gaussian_weights[42] * image_data[offset-3];
        val += gaussian_weights[43] * image_data[offset-2];
        val += gaussian_weights[44] * image_data[offset-1];
        val += gaussian_weights[45] * image_data[offset];
        val += gaussian_weights[46] * image_data[offset+1];
        val += gaussian_weights[47] * image_data[offset+2];
        val += gaussian_weights[48] * image_data[offset+3];
#endif

        image_data_gaussian[h_im * image_gaussian_pitch + w_im] = val;
    }

}



__global__ void imgaussian_GPU_v2(int n_threads,
                                  int height, int width,
                                  int roiheight, int roiwidth,
                                  unsigned char* image_data,
                                  unsigned char* image_data_gaussian,
                                  float* gaussian_weights)
{
    int index = blockDim.x * blockIdx.x +  threadIdx.x;

    if(index < n_threads)
    {
        if(threadIdx.x < 28)   // per block only 3 locations
        {
            __shared__ float shared_sum[32];

            const int idx = (threadIdx.x / 7) +  3 * blockIdx.x;
            const int idx_w = (threadIdx.x % 7);

            const int idx_h = (threadIdx.x / 7);

            //            const int eff_h = height - 2*BORDER_SKIP;
            const int eff_w = width - 2*3;

            int h = idx / eff_w;
            int w = idx % eff_w;

            int h_im = h + 3;
            int w_im = w + 3;

            if( h_im < height && w_im < width)
            {
                int count =0;
                int row_offset = 0;
                int col_offset = 0;

                col_offset =  w_im - 3 + idx_w;

                float sum = 0;
                for(int i=-3;i<3;i++)
                {
                    row_offset = (h_im + i) * width;

                    sum += gaussian_weights[count*7+idx_w] * image_data[row_offset + col_offset];
                    count++;
                }

                shared_sum[threadIdx.x] = sum;

                __syncthreads();

                if(threadIdx.x % 7 == 0)
                {
                    float sum = 0;
                    for(int i=0;i<7;i++)
                        sum += shared_sum[idx_h*7+i];

                    image_data_gaussian[h_im * width + w_im] = sum;

                }
            }

        }
    }
}



void ORB_GPU::compute_gaussian(int height, int width,
                               unsigned char* image_data_gpu,
                               int image_pitch,
                               unsigned char* image_data_gaussian_gpu,
                               int image_gaussian_pitch,
                               float* gaussian_weights_gpu,
                               hipStream_t& hip_stream)
{
    {
        // 3 is (7-1)/2 for gaussian kernel of 7
        // we dont process border because they are not involved in anycomputation
        // in orb because of BORDER_SKIP and BORDER_OFFSET
        int roiheight = (height - 2 * BORDER_SKIP);
        int roiwidth  = (width - 2 * BORDER_SKIP);

        int n_threads =  roiheight * roiwidth;

        int CUDA_NUM_BLOCKS = (n_threads - 1) / CUDA_NUM_THREADS_PER_BLOCK  + CUDA_NUM_THREADS_PER_BLOCK;


        imgaussian_GPU<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS_PER_BLOCK, 0, hip_stream>>>(
                                                                                          n_threads,
                                                                                          height, width,
                                                                                          roiheight, roiwidth,
                                                                                          image_data_gpu,
                                                                                          image_pitch,
                                                                                          image_data_gaussian_gpu,
                                                                                          image_gaussian_pitch,
                                                                                          gaussian_weights_gpu);


        // cz 3 per block

        //        int CUDA_NUM_BLOCKS = ((roiheight * roiwidth - 1) / 3)  + 1;
        //        int n_threads =  CUDA_NUM_BLOCKS * CUDA_NUM_THREADS_PER_BLOCK_V2;


        //        imgaussian_GPU_v2<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS_PER_BLOCK_V2, 0, hip_stream>>>(
        //                                                                                             n_threads,
        //                                                                                             height, width,
        //                                                                                             roiheight, roiwidth,
        //                                                                                             image_data_gpu,
        //                                                                                             image_data_gaussian_gpu,
        //                                                                                             gaussian_weights_gpu);

    }
}




__global__ void imgaussian_GPU_unroll(int n_threads,
                                      int height, int width,
                                      int roiheight, int roiwidth,
                                      unsigned char* image_data,
                                      float* gaussian_weights,
                                      unsigned char* gaussian_unrolling)
{
    int index = blockDim.x * blockIdx.x +  threadIdx.x;

    if(index < n_threads)
    {
        int h = (index / 7) / roiwidth;
        int w = (index / 7) % roiwidth;

        int h_im = h + 3;
        int w_im = w + 3;

        int kernel_loc_w = index % 7;
        int col_offset = w_im + kernel_loc_w - 3;

        float val = 0;
        int count = 0;

        for(int i=-3;i<=3;i++)
        {
            int row_offset = (h_im + i) * width;
            val += gaussian_weights[count * 7 + kernel_loc_w] * image_data[row_offset + col_offset];
            count++;
        }

        const int spatial_offset = roiwidth * roiheight;
        const int spatial_w = index / 7;

        gaussian_unrolling[kernel_loc_w * spatial_offset + spatial_w] = val;
    }

}


__global__ void imgaussian_GPU_reduce(int n_threads,
                                      int height, int width,
                                      int roiheight, int roiwidth,
                                      unsigned char* gaussian_unrolling,
                                      unsigned char* image_data_gaussian)
{
    int index = blockDim.x * blockIdx.x +  threadIdx.x;

    if(index < n_threads)
    {
        const int h = index / roiwidth;
        const int w = index % roiwidth;

        const int h_im = h + 3;
        const int w_im = w + 3;

        const int col_offset = index;

        float val = 0;

        for(int i=0;i<7;i++)
        {
            int row_offset = i * n_threads;
            val +=  gaussian_unrolling[row_offset + col_offset];
        }

        image_data_gaussian[h_im * width + w_im] = val;
    }
}


//__global__ void imgaussian_GPU_unroll(int n_threads,
//                                      int height, int width,
//                                      int roiheight, int roiwidth,
//                                      unsigned char* image_data,
//                                      float* gaussian_weights,
//                                      unsigned char* gaussian_unrolling)
//{
//    int index = blockDim.x * blockIdx.x +  threadIdx.x;

//    if(index < n_threads)
//    {
//        int h = (index / 7) / roiwidth;
//        int w = (index / 7) % roiwidth;

//        int h_im = h + 3;
//        int w_im = w + 3;

//        int kernel_loc_w = index % 7;
//        int col_offset = w_im + kernel_loc_w - 3;

//        float val = 0;
//        int count = 0;

//        for(int i=-3;i<=3;i++)
//        {
//            int row_offset = (h_im + i) * width;
//            val += gaussian_weights[count * 7 + kernel_loc_w] * image_data[row_offset + col_offset];
//            count++;
//        }

//        const int idx = h * roiwidth + w;

//        gaussian_unrolling[idx * 8 + kernel_loc_w] = val;
//    }

//}


//__global__ void imgaussian_GPU_reduce(int n_threads,
//                                      int height, int width,
//                                      int roiheight, int roiwidth,
//                                      unsigned char* gaussian_unrolling,
//                                      unsigned char* image_data_gaussian)
//{
//    int index = blockDim.x * blockIdx.x +  threadIdx.x;

//    if(index < n_threads)
//    {
//        __shared__ int shared_sum[32];

//        const int thread_blkidx = threadIdx.x;

//        const int h = (index/32) / roiwidth;
//        const int w = (index/32) % roiwidth;

//        const int h_im = h + 3;
//        const int w_im = w + 3;

//        shared_sum[thread_blkidx] = gaussian_unrolling[index];

//        int groupsize = 4;

//        for(int i=0;i<3;i++)
//        {
//            if(thread_blkidx < groupsize)
//                shared_sum[thread_blkidx] += shared_sum[thread_blkidx + groupsize];
//        }

//        if(thread_blkidx == 0)
//        {
//            image_data_gaussian[h_im * width + w_im] = shared_sum[0];
//        }

//    }
//}



void ORB_GPU::compute_gaussian_v2(int height, int width,
                                  unsigned char* image_data_gpu,
                                  unsigned char* gaussian_unrolling,
                                  unsigned char* image_data_gaussian_gpu,
                                  float* gaussian_weights_gpu,
                                  hipStream_t& hip_stream)
{


    {
        // 3 is (7-1)/2 for gaussian kernel of 7
        // we dont process border because they are not involved in anycomputation
        // in orb because of BORDER_SKIP and BORDER_OFFSET
        int roiheight = (height - 2 * 3);
        int roiwidth  = (width - 2 * 3);

        int n_threads =  roiheight * roiwidth * 7;

        int CUDA_NUM_BLOCKS = (n_threads -1 ) / CUDA_NUM_THREADS_PER_BLOCK + 1;


        imgaussian_GPU_unroll<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS_PER_BLOCK, 0, hip_stream>>>(
                                                                                                 n_threads,
                                                                                                 height, width,
                                                                                                 roiheight, roiwidth,
                                                                                                 image_data_gpu,
                                                                                                 gaussian_weights_gpu,
                                                                                                 gaussian_unrolling);

        hipStreamSynchronize(hip_stream);
        std::cout << " ---- " << hipGetErrorString(hipGetLastError()) << "\n";

    }


    {
        // 3 is (7-1)/2 for gaussian kernel of 7
        // we dont process border because they are not involved in anycomputation
        // in orb because of BORDER_SKIP and BORDER_OFFSET
        int roiheight = (height - 2 * 3);
        int roiwidth  = (width - 2 * 3);

        int n_threads =  roiheight * roiwidth;

        int CUDA_NUM_BLOCKS = (n_threads -1 ) / CUDA_NUM_THREADS_PER_BLOCK + 1;


        imgaussian_GPU_reduce<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS_PER_BLOCK, 0, hip_stream>>>(
                                                                                                 n_threads,
                                                                                                 height, width,
                                                                                                 roiheight, roiwidth,
                                                                                                 gaussian_unrolling,
                                                                                                 image_data_gaussian_gpu);

        hipStreamSynchronize(hip_stream);

        std::cout << " **** " << hipGetErrorString(hipGetLastError()) << "\n";
    }


    //    {
    //        // 3 is (7-1)/2 for gaussian kernel of 7
    //        // we dont process border because they are not involved in anycomputation
    //        // in orb because of BORDER_SKIP and BORDER_OFFSET
    //        int roiheight = (height - 2 * 3);
    //        int roiwidth  = (width - 2 * 3);

    //        int n_threads =  roiheight * roiwidth * 7;

    //        int CUDA_NUM_BLOCKS = (n_threads -1 ) / CUDA_NUM_THREADS_PER_BLOCK + 1;


    //        imgaussian_GPU_unroll<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS_PER_BLOCK, 0, hip_stream>>>(
    //                                                                                                 n_threads,
    //                                                                                                 height, width,
    //                                                                                                 roiheight, roiwidth,
    //                                                                                                 image_data_gpu,
    //                                                                                                 gaussian_weights_gpu,
    //                                                                                                 gaussian_unrolling);

    //        hipStreamSynchronize(hip_stream);
    //        std::cout << " ---- " << hipGetErrorString(hipGetLastError()) << "\n";

    //    }


    //    {
    //        // 3 is (7-1)/2 for gaussian kernel of 7
    //        // we dont process border because they are not involved in anycomputation
    //        // in orb because of BORDER_SKIP and BORDER_OFFSET
    //        int roiheight = (height - 2 * 3);
    //        int roiwidth  = (width - 2 * 3);

    //        int n_threads =  roiheight * roiwidth * 32;

    //        int CUDA_NUM_BLOCKS = (n_threads -1 ) / CUDA_NUM_THREADS_PER_BLOCK_V2 + 1;


    //        imgaussian_GPU_reduce<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS_PER_BLOCK_V2, 0, hip_stream>>>(
    //                                                                                                 n_threads,
    //                                                                                                 height, width,
    //                                                                                                 roiheight, roiwidth,
    //                                                                                                 gaussian_unrolling,
    //                                                                                                 image_data_gaussian_gpu);

    //        hipStreamSynchronize(hip_stream);

    //        std::cout << " **** " << hipGetErrorString(hipGetLastError()) << "\n";
    //    }



}

}
